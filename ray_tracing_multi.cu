#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hiprand/hiprand_kernel.h>
#include "mpi.h"

typedef struct Vector
{
    float x;
    float y;
    float z;
} Vector;

__device__ inline void vector_add(const Vector V, const Vector U, Vector *result)
{
    result->x = V.x + U.x;
    result->y = V.y + U.y;
    result->z = V.z + U.z;
}

__device__ inline void vector_subtract(const Vector V, const Vector U, Vector *result)
{
    result->x = V.x - U.x;
    result->y = V.y - U.y;
    result->z = V.z - U.z;
}

__device__ inline void vector_multiply(const float t, const Vector V, Vector *result)
{
    result->x = t * V.x;
    result->y = t * V.y;
    result->z = t * V.z;
}

__device__ inline void vector_divide(const float t, const Vector V, Vector *result)
{
    result->x = V.x / t;
    result->y = V.y / t;
    result->z = V.z / t;
}

__device__ inline float vector_dot_product(const Vector V, const Vector U)
{
    return V.x * U.x + V.y * U.y + V.z * U.z;
}

__device__ inline float vector_norm(const Vector V)
{
    return sqrtf(vector_dot_product(V, V));
}

__global__ void ray_tracing(float L_x, float L_y, float L_z, float W_y, float W_max, float C_x, float C_y, float C_z, float R, int ngrid, long nrays, float *d_matrix, unsigned long long *d_total_rays, int rank, int size)
{
    int idx = rank * nrays / size + blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nrays)
        return;

    int total_threads = gridDim.x * blockDim.x;
    long base_work = nrays / size / total_threads;
    long remainder = nrays / size % total_threads;
    long nrays_thread = (idx < remainder) ? base_work + 1 : base_work;

    const Vector L = {L_x, L_y, L_z};
    const Vector C = {C_x, C_y, C_z};

    const float W_max2 = W_max * W_max;
    const float dx = (ngrid - 1) / (2 * W_max);
    const float dz = (ngrid - 1) / (2 * W_max);
    const float view_ray_constant = R * R - vector_dot_product(C, C);
    unsigned long long total = 0;

    float phi, cos_theta, sin_theta, dot_prod, t, b;
    float view_ray_equation = 0;
    int i, j;

    hiprandStateXORWOW_t state;
    hiprand_init(123456789 + idx, 1, 0, &state);
    Vector V, W, I, N, S;
    W.x = W.z = 0;
    W.y = W_y;

    for (long k = 0; k < nrays_thread; ++k)
    {
        while (view_ray_equation <= 0 || W.x * W.x >= W_max2 || W.z * W.z >= W_max2)
        {
            phi = hiprand_uniform(&state) * M_PI;
            cos_theta = 2.0 * hiprand_uniform(&state) - 1.0;
            sin_theta = sqrtf(1 - cos_theta * cos_theta);

            V.x = sin_theta * cosf(phi);
            V.y = sin_theta * sinf(phi);
            V.z = cos_theta;

            vector_multiply(W_y / V.y, V, &W);
            dot_prod = vector_dot_product(V, C);
            view_ray_equation = dot_prod * dot_prod + view_ray_constant;
            total++;
        }

        t = dot_prod - sqrtf(view_ray_equation);
        vector_multiply(t, V, &I);

        vector_subtract(I, C, &N);
        vector_divide(vector_norm(N), N, &N);

        vector_subtract(L, I, &S);
        vector_divide(vector_norm(S), S, &S);

        b = (vector_dot_product(S, N) > 0) ? vector_dot_product(S, N) : 0;
        i = (W.x + W_max) * dx;
        j = (W.z + W_max) * dz;

        atomicAdd(&d_matrix[ngrid * i + j], b);
        view_ray_equation = 0;
    }

    atomicAdd(d_total_rays, total);
}

int main(int argc, char *argv[])
{
    if (argc != 5)
    {
        fprintf(stderr, "Usage: %s <NRAYS> <NGRID> <NBLOCKS> <NTHREADS_PER_BLOCK> \n", argv[0]);
        return EXIT_FAILURE;
    }

    int rank, size;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    float *g_matrix = NULL;
    unsigned long long g_total_rays = 0;

    long NRAYS = atol(argv[1]);
    int NGRID = atoi(argv[2]);
    int NBLOCKS = atoi(argv[3]);
    int NTHREADS_PER_BLOCK = atoi(argv[4]);

    float *d_matrix, *l_matrix = (float *)calloc(NGRID * NGRID, sizeof(float));
    unsigned long long *d_total_rays, *l_total_rays = (unsigned long long *)calloc(1, sizeof(unsigned long long));
    if (rank == 0)
        g_matrix = (float *)calloc(NGRID * NGRID, sizeof(float));

    hipEvent_t start_kernel, stop_kernel;
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    float kernel_time;

    MPI_Barrier(MPI_COMM_WORLD);
    double start = MPI_Wtime();

    assert(hipMalloc((void **)&d_matrix, NGRID * NGRID * sizeof(float)) == hipSuccess);
    assert(hipMalloc((void **)&d_total_rays, sizeof(unsigned long long)) == hipSuccess);

    assert(hipMemset(d_matrix, 0, NGRID * NGRID * sizeof(float)) == hipSuccess);
    assert(hipMemset(d_total_rays, 0, sizeof(unsigned long long)) == hipSuccess);

    hipEventRecord(start_kernel, 0);
    ray_tracing<<<NBLOCKS, NTHREADS_PER_BLOCK>>>(4, 4, -1, 2, 2, 0, 12, 0, 6, NGRID, NRAYS, d_matrix, d_total_rays, rank, size);
    hipDeviceSynchronize();
    hipEventRecord(stop_kernel, 0);

    assert(hipMemcpy(l_matrix, d_matrix, NGRID * NGRID * sizeof(float), hipMemcpyDeviceToHost) == hipSuccess);
    assert(hipMemcpy(l_total_rays, d_total_rays, sizeof(unsigned long long), hipMemcpyDeviceToHost) == hipSuccess);

    MPI_Reduce(l_matrix, g_matrix, NGRID * NGRID, MPI_FLOAT, MPI_SUM, 0, MPI_COMM_WORLD);
    MPI_Reduce(l_total_rays, &g_total_rays, 1, MPI_UNSIGNED_LONG_LONG, MPI_SUM, 0, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);

    if (rank == 0)
    {
        FILE *file = fopen("matrix_cuda.out", "w");

        for (int i = 0; i < NGRID * NGRID; ++i)
            fprintf(file, "%.2lf ", g_matrix[i]);

        fclose(file);
        free(g_matrix);

        double stop = MPI_Wtime();
        double total_time = 1000 * (stop - start);
        hipEventElapsedTime(&kernel_time, start_kernel, stop_kernel);

        printf("\nTotal Time of Execution  : %lf (ms)\n", total_time);
        printf("Kernel Time of Execution : %lf (ms)\n", kernel_time);
        printf("Number of Accepted Rays  : %ld\n", NRAYS);
        printf("Number of Rejected Rays  : %ld\n\n", g_total_rays - NRAYS);
    }

    free(l_matrix);
    free(l_total_rays);
    hipFree(d_matrix);
    hipFree(d_total_rays);

    MPI_Finalize();
    return EXIT_SUCCESS;
}
